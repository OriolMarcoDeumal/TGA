#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include <sys/times.h>
#include <sys/resource.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

using namespace std;

// Funciones del kernel
__global__ void histogram_kernel(unsigned char *input_ptr, int *histogram, int width, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int px = idx / 3;
    int channel = idx % 3;
if (px < width * height && channel == 0) {
    int Y = (int)(16 + 0.25679890625 * input_ptr[px * 3 + 0] + 0.50412890625 * input_ptr[px * 3 + 1] + 0.09790625 * input_ptr[px * 3 + 2]);
    atomicAdd(&histogram[Y], 1);
}
}


__global__ void equalize_kernel(unsigned char *input_ptr, int *histogram_equalized, int width, int height) {
     int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int px = idx / 3;
    int channel = idx % 3;
if (px < width * height && channel == 0) {
    int value_before = input_ptr[idx];
    int value_after = histogram_equalized[value_before];
    input_ptr[idx] = value_after;
}
}

__global__ void ycbcr_kernel(unsigned char *input_ptr, int width, int height, bool toYCbCr) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < width * height * 3) {
        int r = input_ptr[idx + 0];
        int g = input_ptr[idx + 1];
        int b = input_ptr[idx + 2];

        if (toYCbCr) {
            int Y = (int) (16 + 0.25679890625 * r + 0.50412890625 * g + 0.09790625 * b);
            int Cb = (int) (128 - 0.168736 * r - 0.331264 * g + 0.5 * b);
            int Cr = (int) (128 + 0.5 * r - 0.418688 * g - 0.081312 * b);

            input_ptr[idx + 0] = Y;
            input_ptr[idx + 1] = Cb;
            input_ptr[idx + 2] = Cr;
        } else {
            int Y = r;
            int Cb = g;
            int Cr = b;

            int R = max(0, min(255, (int) (Y + 1.402 * (Cr - 128))));
            int G = max(0, min(255, (int) (Y - 0.344136 * (Cb - 128) - 0.714136 * (Cr - 128))));
            int B = max(0, min(255, (int) (Y + 1.772 * (Cb - 128))));

            input_ptr[idx + 0] = R;
            input_ptr[idx + 1] = G;
            input_ptr[idx + 2] = B;
        }
   

}}

// Función para verificar los errores de CUDA
void CheckCudaError(char sms[], int line) {
    hipError_t error;
    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s at line %d\n", sms, line);
        exit(-1);
    }
}

int loadImg(char* fileIN, char* fileOUT) {
  printf("Reading image...\n");
  int channels;
  unsigned char *image = stbi_load(fileIN, &width, &height, &channels, 0);
  if (!image) {
    fprintf(stderr, "Couldn't load image.\n");
    return (-1);
  }
  printf("Image Read. Width : %d, Height : %d, nComp: %d\n", width, height, channels);

  printf("Filtrando\n");
  // Transferir la imagen desde la memoria del sistema a la memoria de la GPU
  unsigned char *d_image;
  hipMalloc((void **)&d_image, width * height * channels * sizeof(unsigned char));
  hipMemcpy(d_image, image, width * height * channels * sizeof(unsigned char), hipMemcpyHostToDevice);

  // Ejecutar el kernel para aplicar el filtro
  dim3 block_dim(256, 1, 1);
  dim3 grid_dim((width * height * channels + block_dim.x - 1) / block_dim.x, 1, 1);
  eq_GPU<<<grid_dim, block_dim>>>(d_image, width, height, channels);

  // Transferir la imagen resultante desde la memoria de la GPU a la memoria del sistema
  hipMemcpy(image, d_image, width * height * channels * sizeof(unsigned char), hipMemcpyDeviceToHost);

  printf("Escribiendo\n");
  // ESCRITURA DE LA IMAGEN EN SECUENCIAL
  stbi_write_png(fileOUT, width, height, channels, image, 0);

  // Liberar la memoria de la GPU
  hipFree(d_image);

  // Liberar la memoria de la imagen
  stbi_image_free(image);

  return (0);
}

__global__ void eq_GPU(unsigned char *input_ptr, int width, int height, int channels)
{
    // Se define el histograma como una variable compartida entre los hilos de un bloque
    __shared__ int histogram[256];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Se inicializa el histograma en 0
    if (threadIdx.x < 256) {
        histogram[threadIdx.x] = 0;
    }

    __syncthreads();

    // Cada hilo procesa un pixel de la imagen
    if (idx < width * height * channels) {
        int Y = (int) (16 + 0.25679890625 * input_ptr[idx + 0] + 0.50412890625 * input_ptr[idx + 1] + 0.09790625 * input_ptr[idx + 2]);

        // Cada hilo aumenta en 1 el valor del histograma en la posición correspondiente
        atomicAdd(&histogram[Y], 1);
    }

    __syncthreads();

    // Se calcula el histograma acumulado
    for (int i = 1; i < 256; i++) {
        histogram[i] += histogram[i - 1];
    }

    __syncthreads();

    // Se normaliza el histograma acumulado
    float normalization_factor = 255.0f / (width * height);
    for (int i = 0; i < 256; i++) {
        histogram[i] = (int)(histogram[i] * normalization_factor + 0.5f);
        histogram[i] = min(histogram[i], 255);
    }

    __syncthreads();

    // Se aplica la ecualización del histograma a la imagen
    if (idx < width * height * channels) {
        int value_before = input_ptr[idx];
        int value_after = histogram[value_before];
        input_ptr[idx] = value_after;
    }
}

  int main(int argc, char** argv) {
    if (argc != 3) {
        cout << "Usage: " << argv[0] << " <input_file> <output_file>" << endl;
        return (-1);
    }

    char* fileIN = argv[1];
    char* fileOUT = argv[2];

    // Cargar imagen
    int width, height, channels;
    unsigned char* image = stbi_load(fileIN, &width, &height, &channels, 0);
    if (!image) {
        fprintf(stderr, "Couldn't load image.\n");
        return (-1);
    }

    // Reservar memoria en la GPU
    unsigned char* d_image;
    hipMalloc((void **)&d_image, width * height * channels * sizeof(unsigned char));
    hipMemcpy(d_image, image, width * height * channels * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Definir dimensiones del grid y del bloque
    dim3 block_dim(256, 1, 1);
    dim3 grid_dim((width * height * channels + block_dim.x - 1) / block_dim.x, 1, 1);

    // Crear histograma
    int* histogram;
    hipMallocManaged(&histogram, 256 * sizeof(int));
    hipMemset(histogram, 0, 256 * sizeof(int));

    // Cambios: Convertir la imagen de RGB a YCbCr
    ycbcr_kernel<<<grid_dim, block_dim>>>(d_image, width, height, /*toYCbCr=*/true);

    // Ejecutar kernel para crear histograma
    histogram_kernel<<<grid_dim, block_dim>>>(d_image, histogram, width, height);

    // Verificar errores de CUDA
    CheckCudaError((char *)"Error creando histograma", __LINE__);

    // Calcular el histograma acumulado
    int* histogram_accumulated;
    hipMallocManaged(&histogram_accumulated, 256 * sizeof(int));
    hipMemset(histogram_accumulated, 0, 256 * sizeof(int));
    int sum = 0;
    for (int i = 0; i < 256; i++) {
        sum += histogram[i];
        histogram_accumulated[i] = sum;
    }

    // Verificar errores de CUDA
    CheckCudaError((char *)"Error calculando histograma acumulado", __LINE__);

    // Crear arreglo de histograma equalizado
    int* histogram_equalized;
    hipMallocManaged(&histogram_equalized, 256 * sizeof(int));
    hipMemset(histogram_equalized, 0, 256 * sizeof(int
    for (int i = 0; i < 256; i++) {
        histogram_equalized[i] = (int) (255.0f * histogram_accumulated[i] / (width * height));
    }

    // Verificar errores de CUDA
    CheckCudaError((char *)"Error creando histograma equalizado", __LINE__);

    // Ejecutar kernel para equalizar la imagen
    equalize_kernel<<<grid_dim, block_dim>>>(d_image, histogram_equalized, width, height);

    // Verificar errores de CUDA
    CheckCudaError((char *)"Error al ejecutar kernel de equalización", __LINE__);

    // Cambios: Convertir la imagen de YCbCr a RGB
    ycbcr_kernel<<<grid_dim, block_dim>>>(d_image, width, height, /*toYCbCr=*/false);

    // Verificar errores de CUDA
    CheckCudaError((char *)"Error al convertir la imagen a RGB", __LINE__);

    // Transferir la imagen de la GPU al CPU
    hipMemcpy(image, d_image, width * height * channels * sizeof(unsigned char), hipMemcpyDeviceToHost);
    // Guardar la imagen resultante
    stbi_write_png(fileOUT, width, height, channels, image, width * channels);

    // Liberar memoria de la GPU y CPU
    hipFree(d_image);
    hipFree(histogram);
    hipFree(histogram_accumulated);
    hipFree(histogram_equalized);
    stbi_image_free(image);

    return 0;
}


